#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <string.h>

__global__ void RS(char* word, char* ans, int len) {
    int idx = blockIdx.x;
    int st = 0;
    for(int i = len; i > len-idx; i--) {
        st += i;
    }
    for(int i = st; i < st + len - idx; i++) {
        ans[i] = word[i-st];
    }
}

int main() {
    char *d_word, *word, *d_ans, *ans;
    word = (char*) malloc(50 * sizeof(char));
    printf("\nEnter the word:\n");
    scanf("%s", word);
    int n = (strlen(word) + 1) * sizeof(char);
    int anslen = ((n*(n-1)/2) + 1) * sizeof(char);
    ans = (char*) malloc(anslen);
    hipMalloc((void**) &d_word, n);
    hipMalloc((void**) &d_ans, anslen);
    hipMemcpy(d_word, word, n, hipMemcpyHostToDevice);
    RS<<<n-1,1>>>(d_word, d_ans, n-1);
    hipMemcpy(ans, d_ans, anslen, hipMemcpyDeviceToHost);
    ans[anslen-1] = '\0';
    printf("\nFinal Word: %s\n", ans);
    hipFree(d_word);
    hipFree(d_ans);
    free(word);
    free(ans);
    return 0;
}